//#include <stdio.h>

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void HelloWorld(void)
{
    printf("Hello World from GPU \n");
}

int main(void)
{
    HelloWorld <<<1, 10>>>();
    hipDeviceReset();
    return 0;
}